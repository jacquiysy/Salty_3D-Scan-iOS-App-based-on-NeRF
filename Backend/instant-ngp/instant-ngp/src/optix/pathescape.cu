#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   pathescape.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  Minimal optix program.
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/random_val.cuh>
#include <optix.h>

#include "pathescape.h"

using namespace tcnn;

NGP_NAMESPACE_BEGIN

extern "C" {
	__constant__ PathEscape::Params params;
}

struct Onb {
	inline __device__ Onb(const vec3& normal) {
		m_normal = normal;

		if (fabs(m_normal.x) > fabs(m_normal.z)) {
			m_binormal.x = -m_normal.y;
			m_binormal.y =  m_normal.x;
			m_binormal.z =  0;
		} else {
			m_binormal.x =  0;
			m_binormal.y = -m_normal.z;
			m_binormal.z =  m_normal.y;
		}

		m_binormal = normalize(m_binormal);
		m_tangent = cross(m_binormal, m_normal);
	}

	inline __device__ void inverse_transform(vec3& p) const {
		p = p.x*m_tangent + p.y*m_binormal + p.z*m_normal;
	}

	vec3 m_tangent;
	vec3 m_binormal;
	vec3 m_normal;
};

extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	vec3 query_point = params.ray_origins[idx.x];

	static constexpr uint32_t N_PATHS = 32;
	static constexpr uint32_t N_BOUNCES = 4;

	default_rng_t rng;
	rng.advance(idx.x * 4 * N_PATHS * N_BOUNCES);
	uint32_t n_escaped = 0;
	for (uint32_t i = 0; i < N_PATHS; ++i) {
		vec3 ray_origin = query_point;
		vec3 ray_direction = random_dir(rng);

		for (uint32_t j = 0; j < N_BOUNCES; ++j) {
			// Trace the stab ray against our scene hierarchy
			unsigned int p0;
			optixTrace(
				params.handle,
				to_float3(ray_origin),
				to_float3(ray_direction),
				0.0f,                // Min intersection distance
				1e16f,               // Max intersection distance
				0.0f,                // rayTime
				OptixVisibilityMask(255), // Specify always visible
				OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				0,                   // SBT offset
				1,                   // SBT stride
				0,                   // missSBTIndex
				p0
			);

			// If the ray didn't escape, p0 contains the index of the triangle that was hit.
			if ((int)p0 == -1) {
				// 2 rays escaped. We are definitely outside and the escape was likely not a numerical fluke.
				// Distance doesn't need to be signed.
				if (++n_escaped > 2) {
					return;
				}
				break;
			}

			vec3 N_0;
			float t = params.triangles[p0].ray_intersect(ray_origin, ray_direction, N_0);
			const vec3 N = normalize(faceforward(N_0, ray_direction, N_0));

			// Prevent self-intersections by subtracting 1e-3f from the target distance.
			ray_origin += ray_direction * fmaxf(0.0f, t - 1e-3f);

			ray_direction = random_dir_cosine(rng);
			Onb onb(N);
			onb.inverse_transform(ray_direction);
		}
	}
	params.distances[idx.x] = -params.distances[idx.x];
}

extern "C" __global__ void __miss__ms() {
	optixSetPayload_0((uint32_t)-1);
}

extern "C" __global__ void __closesthit__ch() {
	optixSetPayload_0(optixGetPrimitiveIndex());
}

NGP_NAMESPACE_END
